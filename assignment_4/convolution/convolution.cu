#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "timer.h"

// #define image_height 8192
#define image_height 256
#define image_width 8192
#define filter_height 5
#define filter_width 5
#define filter_width_sep 3

#define border 2
#define border_height 4
#define border_width 4
#define input_height (image_height + border_height)
#define input_width (image_width + border_width)
#define input_height_sep (image_height + 2)
#define input_width_sep (image_width + 2)

#define block_size_x 32
#define block_size_y 32

#define block_size_hor_x 64
#define block_size_hor_y 16

#define block_size_ver_x 1024
#define block_size_ver_y 1
#define DEBUG
#define SEED 1234


#define block_size_x1 16
#define block_size_y1 16



using namespace std;

__constant__ float d_kernel[filter_width*filter_height];


static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error %s \n",hipGetErrorString( result ));
        exit(1);
    }
}
void convolutionSeq(float *output, float *input, float *filter) {
    //for each pixel in the output image

  timer sequentialTime = timer("Sequential");

  sequentialTime.start();

    for (int y=0; y < image_height; y++) {
        for (int x=0; x < image_width; x++) {

            //for each filter weight
            for (int i=0; i < filter_height; i++) {
                for (int j=0; j < filter_width; j++) {
                    output[y*image_width+x] += input[(y+i)*input_width+x+j]
                        * filter[i*filter_width+j];
                }
            }
            output[y*image_width+x] /= 35;
        }
    }
  sequentialTime.stop();
  cout << "convolution (sequential): \t\t" << sequentialTime << endl;

}


__global__ void convolution_kernel_naive(float *output, float *input, float * filter) {
    // global mem address for this thread
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    if(x>input_width-1 || y > input_height -1){
        return;
    }
    //fprintf("error %i:%i",y,x);
    float sum = 0;
    //for each filter weight
    for (int i=0; i < filter_height; i++) {
        for (int j=0; j < filter_width; j++) {
           sum += input[(y+i)*input_width+x+j] * filter[i*filter_width+j];
        }
    }
    output[y*image_width+x] = sum/35;

}

__global__ void convolution_kernel_shared(float *output, float *input) {
    // global mem address for this thread
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if(x<image_width && y < image_height){
        const size_t shared_width = block_size_x1+border_width;
        const size_t shared_height= block_size_y1+ border_height;

        __shared__ float data[shared_width * shared_height];
        //loading top left tile
        data[threadIdx.y*shared_width+threadIdx.x] = input[y*input_width+x];
        // // loading boottom left tile
        if(threadIdx.y < border_width){
            data[(threadIdx.y+blockDim.y)*shared_width+threadIdx.x]
                                    = input[(y+blockDim.y)*input_width+x];
        }
        // loading upper right
        if(threadIdx.x < border_height){
            data[(threadIdx.y)*shared_width+threadIdx.x+blockDim.x]
                                        = input[y*input_width+x+blockDim.x];
        }
        // loading bottom right
        if(threadIdx.x < border_width && threadIdx.y < border_height){
            data[(threadIdx.y+blockDim.y)*shared_width+threadIdx.x+blockDim.x]
                                        = input[(y+blockDim.y)*input_width+x+blockDim.x];
        }

        __syncthreads();
        //fprintf("error %i:%i",y,x);
        float sum = 0;
        //for each filter weight
        for (int i=0; i < filter_height; i++) {
            for (int j=0; j < filter_width; j++) {
            sum += data[(threadIdx.y+i)*shared_width+threadIdx.x+j] * d_kernel[i*filter_width+j];
            }
        }
        output[y*image_width+x] = sum/35;
    }
}




__global__ void
boxfilter_horiz(float * input, float *output, int filter_size, int radius)
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("%i \n",blockDim.x);
    int offset = 0;
    if (blockIdx.x > 0){
        offset = -radius;
    }
    int start = y*input_width + x + offset;
    float t = 0;
    for(int i = 0; i < filter_size; ++ i){
        t+=input[start+i];
    }
    output[start+radius] = t;


    for (int i = 1; i < blockDim.x ; i++)
    {
        t += input[start + i +filter_size -1];
        t -= input[start + i - 1];
        output[start + i+radius] = t;
    }

}

__global__ void
boxfilter_vert(float * input, float *output, int filter_size, int radius)
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    // int row = y*width;
    float t = 0;
    int offset = 0;
    if (blockIdx.y > 0){
        offset = -radius;
    }
    for(int i = 0; i < filter_size; ++ i){
        t+=input[(y+i+ offset)*input_width + x];
    }
    output[(offset + radius)* input_width +x] = t;

    for (int i = 1; i < blockDim.y ; i++)
    {
        t += input[(y+ i + filter_size + offset -1)* input_width + x];
        t -= input[(y+ i -1 + offset)* input_width + x];
        output[(y + i + radius+offset)* input_width+x] = t;
    }

}

__global__ void
merge(float * input,float * input1, float * input2, float *output,int radius){
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idx = (y+ radius) * input_width + x+radius;
    size_t idx_o = y * image_width + x;
    if(y > radius && x > radius && y < image_height+radius && x < image_width + radius)
        output[idx_o]= (input[idx] + input2[idx] + input1[idx]) / 35;

}

// __global__ void convolve(float * input,float * input1, float * input2, float *output,int radius)



double convolutionCUDA(float *output, float *input, float * filter, int type) {
    float *d_input; float *d_output; float * temp; float * res5;float * res3;float *d_filter;

    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");
    int input_size = 0;
    int filter_size = 0;

    if(type > 0){
        input_size = input_height_sep*input_width_sep*sizeof(float);
        filter_size = filter_width_sep * sizeof(float);
    }else{
        input_size = input_height*input_width* sizeof(float);
        filter_size = filter_height*filter_width*sizeof(float);
    }

    // memory allocation

    checkCudaCall(hipMalloc((void **)&d_input, input_size));

    checkCudaCall(hipMalloc((void **)&temp, input_size));

    checkCudaCall(hipMalloc((void **)&res5, input_size));

    checkCudaCall(hipMalloc((void **)&res3, input_size));

    checkCudaCall(hipMalloc((void **)&d_output, image_height*image_width*sizeof(float)));
    checkCudaCall(hipMalloc((void **)&d_filter, filter_size));


    memoryTime.start();
    // host to device

    checkCudaCall(hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice));

    // zero the result array
    checkCudaCall(hipMemset(d_output, 0, image_height*image_width*sizeof(float)));

    if(type == -1){
         checkCudaCall(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), filter, filter_size));

    }
    if(type == 0){
        checkCudaCall(hipMemcpy(d_filter, filter, filter_size, hipMemcpyHostToDevice));
    }

    if(type > 0){
        checkCudaCall(hipMemset(temp, 0, input_size));
        checkCudaCall(hipMemset(res3, 0, input_size));
        checkCudaCall(hipMemset(res5, 0, input_size));
    }



    memoryTime.stop();

    //measure the GPU function
    kernelTime.start();

    if(type == 0){
        dim3 threads(block_size_x, block_size_y);
        dim3 grid(int(ceilf(image_width/(float)threads.x)),
                    int(ceilf(image_height/(float)threads.y)) );
        convolution_kernel_naive<<<grid, threads>>>(d_output, d_input,d_filter);
    }else if(type == -1){
        dim3 threads(block_size_x1, block_size_y1);
        dim3 grid(int(ceilf(image_width/(float)threads.x)),
                    int(ceilf(image_height/(float)threads.y)) );
        convolution_kernel_shared<<<grid, threads>>>(d_output, d_input);
    }else if(type > 0){
        dim3 threads_h(block_size_hor_x,block_size_hor_y);
        dim3 grid_h( int(ceilf(image_width/(float)threads_h.x)),
                    int(ceilf(image_height/(float)threads_h.y)));


        dim3 threads_v(block_size_ver_x,block_size_ver_y);
        dim3 grid_v(int(ceilf(image_width/(float)threads_v.x)),
                    int(ceilf(image_height/(float)threads_v.y)));

        boxfilter_horiz<<<grid_h, threads_h>>>(temp, d_input,5,2);
        boxfilter_vert<<<grid_v, threads_v>>>(res5, temp,5,2);

        // apply 3x3 box filter
        boxfilter_horiz<<<grid_h, threads_h>>>(temp, d_input,3,1);
        boxfilter_vert<<<grid_v, threads_v>>>(res3, temp,3,1);

        // merge result
        threads_v= dim3(32,32);
        grid_v = dim3(int(ceilf(input_width/(float)threads_v.x)),
                        int(ceilf(input_height/(float)threads_v.y)));
        merge<<<grid_v, threads_v>>>(d_input,res5,res3,d_output, 2);
    }
    hipDeviceSynchronize();
    kernelTime.stop();


    //check to see if all went well
    checkCudaCall(hipGetLastError());

    //copy the result back to host memory
    memoryTime.start();
    checkCudaCall(hipMemcpy(output, d_output, image_height*image_width*sizeof(float),hipMemcpyDeviceToHost));
    memoryTime.stop();


    checkCudaCall(hipFree(d_input));
    checkCudaCall(hipFree(d_output));
     checkCudaCall(hipFree(temp));
    checkCudaCall(hipFree(res5));
    checkCudaCall(hipFree(res3));
    checkCudaCall(hipFree(d_filter));


#ifdef DEBUG
    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;
#endif
    return kernelTime.getTimeInSeconds() + memoryTime.getTimeInSeconds();
}

int compare_arrays(float *a1, float *a2, int n) {
    int errors = 0;
    int print = 0;

    for (int i=0; i<n; i++) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n",i,a1[i],a2[i]);
            }
        }

        float diff = (a1[i]-a2[i])/a1[i];
        if (diff > 1e-6f) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n",i,a1[i],a2[i],diff);
            }
        }

    }

    return errors;
}

void testArrays(float *a1, float *a2)
{
    int errors=0;
    errors += compare_arrays(a1, a2, image_height*image_width);
    if (errors > 0) {
        printf("TEST FAILED! %d errors!\n", errors);
    } else {
        printf("TEST PASSED!\n");
    }
}

void runExperiment(float *seq,float *output, float *input, float *filter, int type){
    size_t iterations = 10;
    double time_sum = 0;
    for(size_t i = 0 ; i < iterations;++i){
        time_sum += convolutionCUDA(output, input, filter,type);
    }
    printf("time: %e \n'",time_sum / iterations);
#ifdef DEBUG
    testArrays(seq,output);
#endif
}
int main() {
    int i,j;

    //allocate arrays and fill them
    float *input = (float *) malloc(input_height * input_width * sizeof(float));
    float *input_sep = (float *) malloc(input_height_sep * input_width_sep * sizeof(float));
    float *output1 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output2 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output3 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output4 = (float *) calloc(image_height * image_width, sizeof(float));

    float * filter = (float *) calloc(filter_height * filter_width , sizeof(float));;


    for (i=0; i< input_height * input_width; i++) {
        input[i] = (float) (i % SEED);
    }
    for(i = 0; i < input_height;++i){
        for(j=0;j<input_width;++j){
            if(i> 0 && i < image_height+1 && j> 0 && j< image_width+1 ){
                 input_sep[(i-1)*(input_width_sep)+j-1] = input[i*input_width + j];
            }
        }
    }


//THis is specific for a W==H smoothening filteri, where W and H are odd.
    for (i=0; i<filter_height * filter_width; i++) {
      filter[i] = 1.0;
    }

    for (i=filter_width+1; i<(filter_height - 1) * filter_width; i++) {
        if (i % filter_width > 0 && i % filter_width < filter_width-1)
            filter[i]+=1.0;
    }

    filter[filter_width*filter_height/2]=3.0;

    for(i = 0; i < filter_height * filter_width; ++i){
        if(i % filter_width == 0)
            printf("\n");
        printf("%e ",filter[i]);

    }
    printf("\n");




//end initialization

    //measure the CPU function
    convolutionSeq(output1, input, filter);
    //measure the GPU function
    printf("DUMMY:\n");
   runExperiment(output1,output2, input, filter,0);

   printf("Shared:\n");
   runExperiment(output1,output3, input, filter,-1);


//    printf("separated:\n");
//     runExperiment(output1,output4, input, filter,1);






    free(input);
    free(input_sep);
    free(filter);
    free(output1);
    free(output2);
    free(output3);
    free(output4);

    return 0;
}


