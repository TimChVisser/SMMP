#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(cudaMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(cudaGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda Error \n");
	exit(1);
    }
}

__global__ void vectorAddKernel(int* deviceA, int* deviceResult) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
// insert operation here
    deviceResult[i] = deviceA[i];
}
extern "C"
void histogram(int *v, long n){
  int* deviceIn, *deviceOut;
  int threadBlockSize=256;
  int result[256];
  checkCudaCall(hipMalloc((void **) &deviceIn, n * sizeof(int)));
    if (deviceIn == NULL) {
        printf("Error in cudaMalloc! \n");
        return;
    }
  checkCudaCall(hipMalloc((void **) &deviceOut, n * sizeof(int)));
    if (deviceOut == NULL) {
        checkCudaCall(hipFree(deviceIn));
        printf("Error in cudaMalloc! \n");
        return;
    }


    checkCudaCall(hipMemcpy(deviceIn, v, n * sizeof(int), hipMemcpyDeviceToHost));
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceIn, deviceOut);
    hipDeviceSynchronize();
    checkCudaCall(hipMemcpy(result, deviceOut, n * sizeof(int), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceIn));
    checkCudaCall(hipFree(deviceOut));

}



