#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "timer.h"

#define image_height 8192
#define image_width 1024
#define filter_height 5
#define filter_width 5
#define filter_width_sep 3

#define border 2
#define border_height 4
#define border_width 4
#define input_height (image_height + border_height)
#define input_width (image_width + border_width)
#define input_height_sep (image_height + 2)
#define input_width_sep (image_width + 2)

#define block_size_x 32
#define block_size_y 32

#define block_size_hor_x 32
#define block_size_hor_y 8

#define block_size_ver_x 8
#define block_size_hor_y 32
// #define DEBUG
#define SEED 1234


#define block_size_x1 16
#define block_size_y1 16

using namespace std;

__constant__ float d_kernel[filter_width*filter_height];

void convolutionSeq(float *output, float *input, float *filter) {
    //for each pixel in the output image

  timer sequentialTime = timer("Sequential");

  sequentialTime.start();

    for (int y=0; y < image_height; y++) {
        for (int x=0; x < image_width; x++) {

            //for each filter weight
            for (int i=0; i < filter_height; i++) {
                for (int j=0; j < filter_width; j++) {
                    output[y*image_width+x] += input[(y+i)*input_width+x+j]
                        * filter[i*filter_width+j];
                }
            }
            output[y*image_width+x] /= 35;
        }
    }
  sequentialTime.stop();
  cout << "convolution (sequential): \t\t" << sequentialTime << endl;

}


__global__ void convolution_kernel_naive(float *output, float *input, float * filter) {
    // global mem address for this thread
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    // if(x>input_width-1 || y > input_height -1){
    //     return;
    // }
    //fprintf("error %i:%i",y,x);
    float sum = 0;
    //for each filter weight
    for (int i=0; i < filter_height; i++) {
        for (int j=0; j < filter_width; j++) {
           sum += input[(y+i)*input_width+x+j] * filter[i*filter_width+j];
        }
    }
    output[y*image_width+x] = sum/35;

}

__global__ void convolution_kernel_shared(float *output, float *input) {
    // global mem address for this thread
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    // if(x<image_width-1 && y < image_height-1){
        const size_t shared_width = block_size_x1+border_width;
        const size_t shared_height= block_size_y1+ border_height;

        __shared__ float data[shared_width * shared_height];
        //loading top left tile
        data[threadIdx.y*shared_width+threadIdx.x] = input[y*input_width+x];
        // // loading boottom left tile
        if(threadIdx.y < border_width){
            data[(threadIdx.y+blockDim.y)*shared_width+threadIdx.x]
                                    = input[(y+blockDim.y)*input_width+x];
        }
        // loading upper right
        if(threadIdx.x < border_height){
            data[(threadIdx.y)*shared_width+threadIdx.x+blockDim.x]
                                        = input[y*input_width+x+blockDim.x];
        }
        // loading bottom right
        if(threadIdx.x < border_width && threadIdx.y < border_height){
            data[(threadIdx.y+blockDim.y)*shared_width+threadIdx.x+blockDim.x]
                                        = input[(y+blockDim.y)*input_width+x+blockDim.x];
        }

        __syncthreads();
        //fprintf("error %i:%i",y,x);
        float sum = 0;
        //for each filter weight
        for (int i=0; i < filter_height; i++) {
            for (int j=0; j < filter_width; j++) {
            sum += data[(threadIdx.y+i)*shared_width+threadIdx.x+j] * d_kernel[i*filter_width+j];
            }
        }
        output[y*image_width+x] = sum/35;
    // }
}




// __global__ void convolution_kernel_horizontal(float *output, float *input) {
//     // global mem address for this thread
//     const int y = blockIdx.y * blockDim.y + threadIdx.y;
//     const int x = blockIdx.x * blockDim.x + threadIdx.x;
//     const size_t kernel_half = 2;
//     const size_t kernel_size = 5;
//     const size_t width = block_size_hor_x + kernel_size;
//     __shared__ float data[block_size_hor_y * width];
//      //loading top left row tile
//     data[threadIdx.x] = input[y*input_width+x];
//     //loading top right row tile
//     if(threadIdx.x < kernel_size){
//         data[threadIdx.x+blockDim.x] = input[y*input_width+x + blockDim.x];
//     }
//     __syncthreads();
//     float sum = 0;
//     //for each filter weight
//     int start = y*input_width_sep+x;
//     for (int j=0; j < kernel_size; j++) {
//         sum += data[j];

//     }
//     output[y*input_width_sep+x] = sum;

// }

// __global__ void convolution_kernel_vertical(float *output, float *input) {
//     // global mem address for this thread
//     const int y = blockIdx.y * blockDim.y + threadIdx.y;
//     const int x = blockIdx.x * blockDim.x + threadIdx.x;
//     const size_t kernel_half = 2;
//     const size_t kernel_size = 5;

//     if(x>input_width_sep-1 || y > input_height -1){
//         return;
//     }
//      __shared__ float data[blockDim.y + kernel_size];
//     //loading top column tile
//     data[threadIdx.y] = input[y*input_width+x];
//     //loading botom column tile
//     if(threadIdx.x < kernel_size){
//         data[threadIdx.y+blockDim.y] = input[y*input_width+x + blockDim.x];
//     }
//     float sum = 0;
//     //for each filter weight
//     for (int i=0; i < filter_width_sep; i++) {
//         sum += input[(y+i)*input_width_sep+x] * d_kernelS[i];
//     }
//     output[y*image_width+x] = sum;

// }

// __global__ void convolution_kernel_horizontal_shared(float *output, float *input) {
//     // global mem address for this thread
//     const int y = blockIdx.y * blockDim.y + threadIdx.y;
//     const int x = blockIdx.x * blockDim.x + threadIdx.x;

//     __shared__ float data[block_size_hor+2];
//     data[blockIdx.x] = input[y*input_width+x];
//     if(blockIdx.x > blockDim.x -2)
//     {
//         int stride = blockDim.x - blockIdx.x
//         data[blockIdx.x +  stride] =  input[y*input_width+x + stride];
//     }
//     __syncthreads();

//     float sum = 0;
//     //for each filter weight
//     if(blockIdx.x < blockDim.x -1 || x > inpu_width)
//     int i = 0;
//     for (int j=0; j < filter_width; j++) {
//         sum += data[(y+i)*input_width+x+j] * d_kernelS[j];
//     }
//     output[y*input_width+x] = sum;

// }

// __device__ void
// d_boxfilter_x(float * input, float *output, int width, int height, int radius)
// {
//     float scale = 1.0f / (float)((r * 2) + 1);

//     float t;
//     // do left edge
//     t = input[0] * r;

//     for (int x = 0; x < w ; x++)
//     {
//         t += id[x + r];
//         t -= id[x - r - 1];
//         od[x] = t * scale;
//     }

// }

// // process column
// __device__ void
// d_boxfilter_y(float *id, float *od, int w, int h, int r)
// {
//     const int y = blockIdx.y * blockDim.y + threadIdx.y;
//     const int x = blockIdx.x * blockDim.x + threadIdx.x;

//     float scale = 1.0f / (float)((r << 1) + 1);

//     float t;
//     // do left edge
//     t = id[0] * r;

//     for (int y = 0; y < (r + 1); y++)
//     {
//         t += id[y * w];
//     }

//     od[0] = t * scale;

//     for (int y = 1; y < (r + 1); y++)
//     {
//         t += id[(y + r) * w];
//         t -= id[0];
//         od[y * w] = t * scale;
//     }

//     // main loop
//     for (int y = (r + 1); y < (h - r); y++)
//     {
//         t += id[(y + r) * w];
//         t -= id[((y - r) * w) - w];
//         od[y * w] = t * scale;
//     }

//     // do right edge
//     for (int y = h - r; y < h; y++)
//     {
//         t += id[(h-1) * w];
//         t -= id[((y - r) * w) - w];
//         od[y * w] = t * scale;
//     }
// }

// __global__ void
// d_boxfilter_x_global(float *id, float *od, int w, int h, int r)
// {
//     unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;
//     d_boxfilter_x(&id[y * w], &od[y * w], w, h, r);
// }

// __global__ void
// d_boxfilter_y_global(float *id, float *od, int w, int h, int r)
// {
//     unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
//     d_boxfilter_y(&id[x], &od[x], w, h, r);
// }


double convolutionCUDA(float *output, float *input, float * filter, int type) {
    float *d_input; float *d_output; float * d_output2;float *d_filter;
    hipError_t err;
    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");
    int input_size = 0;
    int filter_size = 0;

    if(type > 0){
        input_size = input_height_sep*input_width_sep*sizeof(float);
        filter_size = filter_width_sep * sizeof(float);
    }else{
        input_size = input_height*input_width* sizeof(float);
        filter_size = filter_height*filter_width*sizeof(float);
    }

    // memory allocation

    err = hipMalloc((void **)&d_input, input_size);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString( err )); }

    err = hipMalloc((void **)&d_output2, input_size);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_output2: %s\n",    hipGetErrorString( err )); }

    err = hipMalloc((void **)&d_output, image_height*image_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString( err )); }

    err = hipMalloc((void **)&d_filter, filter_size);


    memoryTime.start();
    // host to device

    err = hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy host to device input: %s\n", hipGetErrorString( err ));  }

    // zero the result array
    err = hipMemset(d_output, 0, image_height*image_width*sizeof(float));
    err = hipMemset(d_output2, 0, image_height*image_width*sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemset output: %s\n", hipGetErrorString( err ));  }

    if(type == -1){
         err = hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), filter, filter_size);
        if (err != hipSuccess) { fprintf(stderr, "Error in cudaSimbolKernel output: %s\n", hipGetErrorString( err ));  }
    }
    if(type == 0){
        err = hipMemcpy(d_filter, filter, filter_size, hipMemcpyHostToDevice);
        if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy host to device filter: %s\n", hipGetErrorString( err ));  }
    }



    memoryTime.stop();

    //measure the GPU function
    kernelTime.start();

    if(type == 0){
        dim3 threads(block_size_x, block_size_y);
        dim3 grid(int(ceilf(image_width/(float)threads.x)),
                    int(ceilf(image_height/(float)threads.y)) );
        convolution_kernel_naive<<<grid, threads>>>(d_output, d_input,d_filter);
    }else if(type == -1){
        dim3 threads(block_size_x1, block_size_y1);
        dim3 grid(int(ceilf(image_width/(float)threads.x)),
                    int(ceilf(image_height/(float)threads.y)) );
        convolution_kernel_shared<<<grid, threads>>>(d_output, d_input);
    }else if(type == -2){
        dim3 threads(5, 6);
        dim3 grid(int(ceilf(image_width/(float)threads.x)),
                    int(ceilf(image_height/(float)threads.y)) );
        convolution_kernel_shared<<<grid, threads>>>(d_output, d_input);
    }else if(type == -3){
        dim3 threads(8, 8);
        dim3 grid(int(ceilf(image_width/(float)threads.x)),
                    int(ceilf(image_height/(float)threads.y)) );
        convolution_kernel_shared<<<grid, threads>>>(d_output, d_input);
    }else if(type > 0){
        // dim3 threads_h(block_size_hor,1);
        // dim3 grid_h( int(ceilf(image_width/(float)threads_h.x)), int(image_height));
        // convolution_kernel_horizontal<<<grid_h, threads_h>>>(d_output2, d_input);

        // dim3 threads_v(1,block_size_ver);
        // dim3 grid_v(int(image_width),int(ceilf(image_height/(float)threads_v.y)));
        // convolution_kernel_vertical<<<grid_v, threads_v>>>(d_output, d_output2);
    }
    hipDeviceSynchronize();
    kernelTime.stop();


    //check to see if all went well
    err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "Error during kernel launch convolution_kernel: %s\n", hipGetErrorString( err )); }

    //copy the result back to host memory
    memoryTime.start();
    err = hipMemcpy(output, d_output, image_height*image_width*sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemcpy device to host output: %s\n", hipGetErrorString( err )); }

    err = hipFree(d_input);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString( err )); }
    err = hipFree(d_output);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString( err )); }
     err = hipFree(d_output2);
    if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString( err )); }

    if(type == 0){
        err = hipFree(d_filter);
        if (err != hipSuccess) { fprintf(stderr, "Error in freeing d_filter: %s\n", hipGetErrorString( err )); }
    }

#ifdef DEBUG
    cout << "convolution (kernel): \t\t" << kernelTime << endl;
    cout << "convolution (memory): \t\t" << memoryTime << endl;
#endif
    return kernelTime.getTimeInSeconds() + memoryTime.getTimeInSeconds();
}

int compare_arrays(float *a1, float *a2, int n) {
    int errors = 0;
    int print = 0;

    for (int i=0; i<n; i++) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n",i,a1[i],a2[i]);
            }
        }

        float diff = (a1[i]-a2[i])/a1[i];
        if (diff > 1e-6f) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t rel_error=\t %10.7e\n",i,a1[i],a2[i],diff);
            }
        }

    }

    return errors;
}

void testArrays(float *a1, float *a2)
{
    int errors=0;
    errors += compare_arrays(a1, a2, image_height*image_width);
    if (errors > 0) {
        printf("TEST FAILED! %d errors!\n", errors);
    } else {
        printf("TEST PASSED!\n");
    }
}

void runExperiment(float *seq,float *output, float *input, float *filter, int type){
    size_t iterations = 100;
    double time_sum = 0;
    for(size_t i = 0 ; i < iterations;++i){
        time_sum += convolutionCUDA(output, input, filter,type);
    }
    printf("time: %e \n'",time_sum / iterations);
#ifdef DEBUG
    testArrays(seq,output);
#endif
}
int main() {
    int i,j;
    hipError_t err;

    //allocate arrays and fill them
    float *input = (float *) malloc(input_height * input_width * sizeof(float));
    float *input_sep = (float *) malloc(input_height_sep * input_width_sep * sizeof(float));
    float *output1 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output2 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output3 = (float *) calloc(image_height * image_width, sizeof(float));
    float *output4 = (float *) calloc(image_height * image_width, sizeof(float));

    float * filter = (float *) calloc(filter_height * filter_width , sizeof(float));;


    for (i=0; i< input_height * input_width; i++) {
        input[i] = (float) (i % SEED);
    }
    for(i = 0; i < input_height;++i){
        for(j=0;j<input_width;++j){
            if(i> 0 && i < image_height+1 && j> 0 && j< image_width+1 ){
                 input_sep[(i-1)*(input_width_sep)+j-1] = input[i*input_width + j];
            }
        }
    }


//THis is specific for a W==H smoothening filteri, where W and H are odd.
    for (i=0; i<filter_height * filter_width; i++) {
      filter[i] = 1.0;
    }

    for (i=filter_width+1; i<(filter_height - 1) * filter_width; i++) {
        if (i % filter_width > 0 && i % filter_width < filter_width-1)
            filter[i]+=1.0;
    }

    filter[filter_width*filter_height/2]=3.0;

    for(i = 0; i < filter_height * filter_width; ++i){
        if(i % filter_width == 0)
            printf("\n");
        printf("%e ",filter[i]);

    }
    printf("\n");




//end initialization

    //measure the CPU function
    convolutionSeq(output1, input, filter);
    //measure the GPU function
    printf("DUMMY:\n");
   runExperiment(output1,output2, input, filter,0);

   printf("5x5:\n");
   runExperiment(output1,output2, input, filter,-1);


//    printf("5x6:\n");
//    runExperiment(output1,output2, input, filter,-2);

//    printf("8x8:\n");
//    runExperiment(output1,output2, input, filter,-3);





    //free(input);
    //free(input_sep);
    free(filter);
    free(output1);
    free(output2);
    free(output3);
    free(output4);

    return 0;
}


